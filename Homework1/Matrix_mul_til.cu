#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 10000
#define M 10000
#define K 10000
#define tile_size 16


__global__ void matrix_mul_shared(float *a, float *b, float *c) {
	
	__shared__ int a_tile[tile_size][tile_size]; 		//define shared memory tile for matrix a
	__shared__ int b_tile[tile_size][tile_size];		//define shared memory tile for matrix b

    int row = blockIdx.y * tile_size + threadIdx.y;	
	int col = blockIdx.x * tile_size + threadIdx.x;	

	float temp = 0.0; //store sum
    int tileIdx; 

	//Load one tile into shared memory
	for (int s = 0; s < gridDim.x; s++) {
		tileIdx = row * K + s * tile_size + threadIdx.x;

		if(tileIdx >= K*K)
			a_tile[threadIdx.y][threadIdx.x] = 0;	//check if K is divisible by tile size for a_tile
		else
			a_tile[threadIdx.y][threadIdx.x] = a[tileIdx];
	

		tileIdx = (s * tile_size + threadIdx.y) * K + col;

		if(tileIdx >= K*K)
			b_tile[threadIdx.y][threadIdx.x] = 0; 	//check if K is divisible by tile size for b_tile
		else
			b_tile[threadIdx.y][threadIdx.x] = b[tileIdx];
			
		__syncthreads(); 
		for (int j = 0; j < tile_size; j++)
			temp += a_tile[threadIdx.y][j] * b_tile[j][threadIdx.x]; //add and multiply

		__syncthreads(); 
		
	}
	
	if(row < K && col < K) 	
		c[row * K + col] = temp; //store the result 
    	
}

//Function to initialize matrices with random values
void randomInit (float *data, int size)	
{
	for (int i = 0; i <  size; i++) 
		for (int j = 0; j < size; j++) 
			*(data + i * size + j) = rand() % 1024; 
}


int main(void)	{
	
	
	float *a, *b, *c; //CPU copies
	float *d_a, *d_b, *d_c;  //GPU copies 
	int matrix_size = N * M * sizeof(float);
	
	hipEvent_t start, stop,start1,stop1,start2,stop2;
	float time,time1,time2;

	//Start the cuda timer
	hipEventCreate(&start);
	hipEventCreate(&start1);
	hipEventCreate(&start2);
	hipEventCreate(&stop);
	hipEventCreate(&stop1);
	hipEventCreate(&stop2);

	//Allocate CPU memory
	a = (float *) malloc(matrix_size);	randomInit(a, N);
	b = (float *) malloc(matrix_size);	randomInit(b, M);
	c = (float *) malloc(matrix_size);

	//Allocate GPU memory 
	hipMalloc((void **) &d_a, matrix_size);
	hipMalloc((void **) &d_b, matrix_size);
	hipMalloc((void **) &d_c, matrix_size);

	//Copy from CPU memory to GPU memory
	hipEventRecord( start1, 0 );
	hipMemcpy( d_a, a, matrix_size, hipMemcpyHostToDevice);
	hipMemcpy( d_b, b, matrix_size, hipMemcpyHostToDevice);
	hipEventRecord( stop1, 0 );
	hipEventSynchronize(stop1);
	hipEventElapsedTime( &time1, start1, stop1 );
	hipEventDestroy( start1 );
	hipEventDestroy( stop1 );

	//Set thread and grid dimensions
	dim3 tBlock(16, 16);
	dim3 Grid((N + 16 - 1)/tBlock.x, (M + 16 - 1)/tBlock.y);

	hipEventRecord( start, 0 );

	//Call kernels
	matrix_mul_shared<<< Grid, tBlock >>> (d_a,d_b,d_c);

	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("GPU Execution Time without memory transfer= %f\n",time);

	//Copy from device to host
	hipEventRecord( start2, 0 );
	hipMemcpy( c, d_c, matrix_size, hipMemcpyDeviceToHost);
	hipEventRecord( stop2, 0 );
	hipEventSynchronize(stop2);
	hipEventElapsedTime( &time2, start2, stop2 );
	hipEventDestroy( start2 );
	hipEventDestroy( stop2 );

	float tTime=time+time1+time2;
	printf("GPU Execution time with memory transfer =%f\n",tTime);
	//free cpu and gpu memory
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}