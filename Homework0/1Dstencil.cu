
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <sys/time.h>

struct timeval stop, start,start1,stop1;

#define R 2
#define B 128
#define N 1000

__global__ void stencil1d(int *in, int *out){
    int gindex=threadIdx.x+(blockIdx.x*blockDim.x) + R;
    int result=0;
    for (int offset = -R; offset <= R ; offset++){
        result += in[gindex + offset];
    }
    out[gindex-R]=result;
}

void random(int *a, int n ){
    int i;
    for (i = 0; i <=n+1; ++i)
     a[i] = rand()%100;
     
 }

int main(void){
    int n;
    int *c_in, *c_out;
    int size= N*sizeof(int);
    n=N+2*R;
    c_in=(int*)malloc(n*size);
    c_out=(int*)malloc(N*size);
    random(c_in,n);
    

    int *d_in,*d_out;
    hipMalloc(&d_in,n*size);
    hipMalloc(&d_out,N*size);

    hipMemcpy(d_in,c_in,n*size,hipMemcpyHostToDevice);

    gettimeofday(&start, NULL);
    stencil1d<<<(N/B-1)/B,B>>>(d_in,d_out);
    gettimeofday(&stop, NULL);
    printf("took %lu us\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);
    hipDeviceSynchronize();
    hipMemcpy(c_out,d_out,n*size,hipMemcpyDeviceToHost);

    free(c_in);
    free(c_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;




}